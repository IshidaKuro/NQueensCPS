#include "hip/hip_runtime.h"
#include <time.h>
#include <stdio.h>
#include <iostream>

#include <vector>
#include <mutex>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include ""

using namespace std;


const int n = 12;

int threadcount = n;
int runs = 10;

struct spile
{
	void Add(vector<int> c)
	{
		lock_guard<std::mutex> guard(mutex);
		s.push_back(c);
	}

	void Clear()
	{
		s.clear();
	}

	bool Contains(vector<int> c)
	{
		lock_guard<std::mutex> guard(mutex);
		if (std::find(s.begin(), s.end(), c) != s.end())
		{
			return true;
		}
		else
		{
			return false;
		}
	}

	int Count()
	{
		return s.size();
	}
private:
	std::mutex mutex;
	vector<vector<int>> s;

};


spile solutions;

 bool CheckDiagonals(int x, int y, vector<int> q)
{

	for (int i = 1; i <= x; i++)
	{
		if (q[x - i] == y - i || q[x - i] == y + i)
		{
			return false;
		}
	}

	return true;

}

int sum(bool q[n], int x)
{
	int s = 0;
	for (int i = 0; i < x; i++)
	{
		if (q[i])
		{
			s++;
		}
	}
	return s;
}


__global__ void kernel(int p)
{
	bool finished = false;
	bool taken[n] = { false }; // store if this row/column has been populated
	bool tried[n][n] = { false }; //store the squares that have been checked
	vector<int> queens;

	queens.push_back(p);
	taken[p] = true;
	for (int i = 0; i < n; i++)
	{
		tried[0][i] = true;
	}
	int x = 1, y = 0; //initialise navigation variables

	//select a random square

	while (!finished)
	{
		y = rand() % n;
		//cout << "(" << x << ", " << y << ")" << endl;
		//if the row is clear and the square has not been tried	
		if (!taken[y] && !tried[x][y])
		{
			tried[x][y] = true; //mark the square as tried


			if (CheckDiagonals(x, y, queens)) //if the diaganols are clear, place the queen and move on to the next column
			{
				queens.push_back(y);
				taken[y] = true;
				x++;
				y = 0;
				if (x == n) //if we are at the end of the board
				{
					if (solutions.Contains(queens))
					{
						taken[queens.back()] = false;

						queens.pop_back();//remove the last queen we placed

						x--; //step back
					}
					else
					{
						//add the solution to the list	
						solutions.Add(queens);
						//reset and find another solution
						queens.clear();
						for (int i = 0; i < n; i++) //y values
						{
							taken[i] = false;
							for (int j = 1; j < n; j++) //x values
							{
								tried[j][i] = false;
							}
						}
						queens.push_back(p);
						taken[p] = true;
						x = 1;
					}
				}



			}
		}

		if (sum(tried[x], n) + sum(taken, n) == n) //if we can not try any more locations
		{
			if (x == 1) { finished = true; }
			taken[queens.back()] = false;

			queens.pop_back();//remove the last queen we placed

			for (int i = 0; i < n; i++)
			{
				tried[x][i] = false;
			}

			x--; //step back

		}



	}

}



int main(int argc, char** argv)
{
	for (int r = 0; r < runs; r++)
	{
		auto start = chrono::system_clock::now();
		/*
			thread threads[n];
			for (int i = 0; i < n; i++)
			{
				threads[i] = thread(Navigate, i);

			}

			for (int i = 0; i < n; i++)
		{
			Navigate(i);
		}

			for (auto& t : threads)
			{
				t.join();
			}


			*/


#pragma omp parallel for num_threads(threadcount)
		for (int i = 0; i < n; i++)
		{
			Navigate(i);
		}


		auto end = chrono::system_clock::now();

		cout << "N=" << n << endl;
		cout << "number of solutions = " << solutions.Count() << endl;
		cout << "time taken: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " milliseconds" << endl;
		solutions.Clear();
	}
	return 0;

}




//check if row is free, and if the if the square has been tried before

//mark position as tried

//check if diaganols are free

//add queen to the solution

//set row as used

//increment the column we are working on


//function to check for diagonal queens

//function to check for queens on same row

//check for column queens


